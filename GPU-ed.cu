#include <iostream>
#include <fstream>


#include "include/ed.cuh"
#include "include/cuda_def.cuh"


int main(int argc, char* argv[]) {

     if (argc != 5){
        std::cout << "call" << argv[0] 
                  << " query.bin subject.bin M N" << std::endl; 
        return 1;
    }

    hipSetDevice(0);                                                     CUERR
    hipDeviceReset();                                                    CUERR

    double *zquery = NULL, *subject = NULL,
           *Subject = NULL, *AvgS = NULL, *StdS = NULL; 

    int M = atoi(argv[3]);
    int N = atoi(argv[4]);

    
    std::cout << "\n= info =====================================" << std::endl;
    std::cout << "|Query| = " << M << "\t"
              << "|Subject| = " << N << "\t" << std::endl;

    // host side memory
    hipHostMalloc(&zquery, sizeof(double)*M);                            CUERR
    hipHostMalloc(&subject, sizeof(double)*N);                           CUERR

    // device side memory
    hipMalloc(&Subject, sizeof(double)*N);                               CUERR
    hipMalloc(&AvgS, sizeof(double)*(N-M+1));                            CUERR
    hipMalloc(&StdS, sizeof(double)*(N-M+1));                            CUERR
    
    // timer events
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    std::cout << "\n= loading data =============================" << std::endl;
    
    hipEventRecord(start, 0);
    
    // read query from file
    std::ifstream qfile(argv[1], std::ios::binary|std::ios::in);
    qfile.read((char *) &zquery[0], sizeof(double)*M);

    // read subject from file
    std::ifstream sfile(argv[2], std::ios::binary|std::ios::in);
    sfile.read((char *) &subject[0], sizeof(double)*N);

    // z-normalize query and envelope
    znormalize(zquery, M);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    std::cout << "Miliseconds to load data: " << time << std::endl;
    
    hipEventRecord(start, 0);
    
    // copy subject to gpu
    hipMemcpy(Subject, subject, sizeof(double)*N, 
               hipMemcpyHostToDevice);                                   CUERR
    // copy query to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(::Czquery), zquery, sizeof(double)*M);              CUERR

    // calculate windowed average and standard deviation of Subject
    avg_std<double>(Subject, AvgS, StdS, M, N);

    // average not needed anymore
    hipFree(AvgS);                                                       CUERR
    
    // calculate best z-normalized Euclidean match
    calculate_ed(Subject, StdS, M, N);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    std::cout << "Miliseconds to find best match: " << time << std::endl;
}

