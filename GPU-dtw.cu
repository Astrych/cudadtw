#include <iostream>
#include <fstream>

#include "include/cuda_def.cuh"
#include "include/cdtw.cuh"

#define EUCLIDEAN (true)
#define MANHATTAN (false)

int main(int argc, char* argv[]) {


     if (argc != 6){
        std::cout << "call" << argv[0] 
                  << " query.bin subject.bin M N P" << std::endl; 
        return 1;
    }

    hipSetDevice(0);                                                     CUERR
    hipDeviceReset();                                                    CUERR

    float *zlower = NULL, *zupper = NULL, *zquery = NULL, *subject = NULL,
          *Subject = NULL, *AvgS = NULL, *StdS = NULL; 

    int M = atoi(argv[3]);
    int N = atoi(argv[4]);
    int W = M*(atoi(argv[5])*0.01);
    
    std::cout << "\n= info =====================================" << std::endl;
    std::cout << "|Query| = " << M << "\t"
              << "|Subject| = " << N << "\t"
              << "window = " << W << std::endl;

    // host side memory
    hipHostMalloc(&zlower, sizeof(float)*M);                             CUERR
    hipHostMalloc(&zupper, sizeof(float)*M);                             CUERR
    hipHostMalloc(&zquery, sizeof(float)*M);                             CUERR
    hipHostMalloc(&subject, sizeof(float)*N);                            CUERR

    // device side memory
    hipMalloc(&Subject, sizeof(float)*N);                                CUERR
    hipMalloc(&AvgS, sizeof(float)*(N-M+1));                             CUERR
    hipMalloc(&StdS, sizeof(float)*(N-M+1));                             CUERR

    // bind subject texture
    hipBindTexture(0, Tsubject, Subject, N*sizeof(float));               CUERR
    
    // timer events
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    std::cout << "\n= loading data =============================" << std::endl;
    
    hipEventRecord(start, 0);
    
    // read query from file
    std::ifstream qfile(argv[1], std::ios::binary|std::ios::in);
    qfile.read((char *) &zquery[0], sizeof(float)*M);

    // read subject from file
    std::ifstream sfile(argv[2], std::ios::binary|std::ios::in);
    sfile.read((char *) &subject[0], sizeof(float)*N);

    // z-normalize query and envelope
    znormalize(zquery, M);
    envelope(zquery, W, zlower, zupper, M);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    std::cout << "Miliseconds to load data: " << time << std::endl;
    
    hipEventRecord(start, 0);
    
    // copy subject to gpu
    hipMemcpy(Subject, subject, sizeof(float)*N, 
               hipMemcpyHostToDevice);                                   CUERR
    // copy query and associated envelopes to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(::Czlower), zlower, sizeof(float)*M);               CUERR
    hipMemcpyToSymbol(HIP_SYMBOL(::Czupper), zupper, sizeof(float)*M);               CUERR
    hipMemcpyToSymbol(HIP_SYMBOL(::Czquery), zquery, sizeof(float)*M);               CUERR

    // calculate windowed average and standard deviation of Subject
    avg_std<double>(Subject, AvgS, StdS, M, N);
    std::cout << "\n= pruning ==================================" << std::endl;
    
    prune_cdtw<EUCLIDEAN>(Subject, AvgS, StdS, M, N, W);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    std::cout << "Miliseconds to find best match: " << time << std::endl;
}

