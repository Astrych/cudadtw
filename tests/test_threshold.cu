#include <iostream>
#include <algorithm>
#include "include/cub_util.cuh"

#define L (100)

int main () {

    int *indices = NULL, *Indices = NULL;
    float *values = NULL, *Values = NULL;
    
    hipHostMalloc(&indices, sizeof(int)*L);
    hipHostMalloc(&values, sizeof(float)*L);
    
    hipMalloc(&Indices, sizeof(int)*L);
    hipMalloc(&Values, sizeof(float)*L);

    for (int i = 0; i < L; ++i)
        indices[i] = values[i] = i;
    std::random_shuffle(values, values+L);

    for (int i = 0; i < L;  ++i)
        std::cout << indices[i] << "\t" << values[i] << std::endl;
    std::cout << "=================================" << std::endl;
    
    hipMemcpy(Indices, indices, sizeof(int)*L, hipMemcpyHostToDevice);
    hipMemcpy(Values, values, sizeof(float)*L, hipMemcpyHostToDevice);
    
    int length;
    threshold(Values, Indices, L, &length, 22.0f);
    
    hipMemcpy(indices, Indices, sizeof(int)*length, hipMemcpyDeviceToHost);
    
    for (int i = 0; i < length; ++i)
        std::cout << i << "\t" << indices[i] << std::endl;

}
