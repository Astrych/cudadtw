#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <algorithm>

#include "include/cdtw.cuh"
#include "include/stats.cuh"

#define EUCLIDEAN (true)
#define MANHATTAN (false)

int main(int argc, char* argv[]) {


     if (argc != 6){
        std::cout << "call" << argv[0] 
                  << " query.bin subject.bin M N P" << std::endl; 
        return 1;
    }

    hipSetDevice(0);                                                     CUERR
    hipDeviceReset();                                                    CUERR

    float *zquery = NULL, *subject = NULL, 
          *avgS = NULL, *stdS = NULL, *cdtw = NULL, 
          *Subject = NULL, *AvgS = NULL, *StdS = NULL, *Cdtw = NULL; 

    int *indices = NULL, *Indices = NULL;

    int M = atoi(argv[3]);
    int N = atoi(argv[4]);
    int W = M*(atoi(argv[5])*0.01);
    
    std::cout << "\n= info =====================================" << std::endl;
    std::cout << "|Query| = " << M << "\t"
              << "|Subject| = " << N << "\t"
              << "window = " << W << std::endl;

    // host side memory
    hipHostMalloc(&zquery, sizeof(float)*M);                             CUERR
    hipHostMalloc(&subject, sizeof(float)*N);                            CUERR
    hipHostMalloc(&avgS, sizeof(float)*(N-M+1));                         CUERR
    hipHostMalloc(&stdS, sizeof(float)*(N-M+1));                         CUERR
    hipHostMalloc(&cdtw, sizeof(float)*(N-M+1));                         CUERR
    hipHostMalloc(&indices, sizeof(int)*(N-M+1));                        CUERR

    // device side memory
    hipMalloc(&Subject, sizeof(float)*N);                                CUERR
    hipMalloc(&AvgS, sizeof(float)*(N-M+1));                             CUERR
    hipMalloc(&StdS, sizeof(float)*(N-M+1));                             CUERR
    hipMalloc(&Cdtw, sizeof(float)*(N-M+1));                             CUERR
    hipMalloc(&Indices, sizeof(int)*(N-M+1));                            CUERR

    // bind subject texture
    hipBindTexture(0, Tsubject, Subject, N*sizeof(float));               CUERR
    
    // timer events
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    std::cout << "\n= loading data =============================" << std::endl;
    
    hipEventRecord(start, 0);
    
    // read query from file
    std::ifstream qfile(argv[1], std::ios::binary|std::ios::in);
    qfile.read((char *) &zquery[0], sizeof(float)*M);

    // read subject from file
    std::ifstream sfile(argv[2], std::ios::binary|std::ios::in);
    sfile.read((char *) &subject[0], sizeof(float)*N);

    // z-normalize query and envelope
    znormalize(zquery, M);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    std::cout << "Miliseconds to load data: " << time << std::endl;
    
    // create a random set of indices
    for (int i = 0; i < N-M+1; ++i)
        indices[i] = i;
    std::random_shuffle(indices, indices+N-M+1);

    hipMemcpy(Indices, indices, sizeof(int)*(N-M+1), 
               hipMemcpyHostToDevice);                                   CUERR

    // copy subject to gpu
    hipMemcpy(Subject, subject, sizeof(float)*N, 
               hipMemcpyHostToDevice);                                   CUERR
    // copy query and associated envelopes to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(::Czquery), zquery, sizeof(float)*M);               CUERR

    // calculate windowed average and standard deviation of Subject
    avg_std<double>(Subject, AvgS, StdS, M, N);

    hipEventRecord(start, 0);
    
    // call gpu dtw
    gpu_cdtw<EUCLIDEAN>(Subject, AvgS, StdS, Cdtw, M, Indices, N-M+1, W); 

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    std::cout << "GPU BLOCK: " << time << std::endl;

    // save result for consistency check
    float *gpucdtw = NULL;
    hipHostMalloc(&gpucdtw, sizeof(float)*(N-M+1));                      CUERR
    hipMemcpy(gpucdtw, Cdtw, sizeof(float)*(N-M+1), 
               hipMemcpyDeviceToHost);                                   CUERR

    hipEventRecord(start, 0);
    
    // call gpu dtw
    gpu_cdtw<EUCLIDEAN>(Subject, AvgS, StdS, Cdtw, M, Indices, N-M+1, W, false); 

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    std::cout << "GPU THRD: " << time << std::endl;

    // copy statistics back to host
    hipMemcpy(avgS, AvgS, sizeof(float)*(N-M+1), hipMemcpyDeviceToHost);CUERR
    hipMemcpy(stdS, StdS, sizeof(float)*(N-M+1), hipMemcpyDeviceToHost);CUERR

    hipEventRecord(start, 0);
    
    // call openmp dtw
    cpu_cdtw <EUCLIDEAN>
    (zquery, subject, avgS, stdS, cdtw, M, indices, N-M+1, W);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    std::cout << "CPU OMP: " << time << std::endl;
    
    hipEventRecord(start, 0);
    
    // call sequential dtw
    cpu_cdtw <EUCLIDEAN>
    (zquery, subject, avgS, stdS, cdtw, M, indices, N-M+1, W, false);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    std::cout << "CPU SEQ: " << time << std::endl;
    
    for (int i = 0; i < N-M+1; ++i)
        if(fabs(gpucdtw[i]-cdtw[i]) > 1E-4)
            std::cout << i << "\t" << gpucdtw[i] << "\t" << cdtw[i] << std::endl;
}
